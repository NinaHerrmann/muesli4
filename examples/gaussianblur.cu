#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2020 Nina Herrmann
 *
 * This software is released under the MIT License.
 * https://opensource.org/licenses/MIT
 */
#include <algorithm>
#include "dm.h"
#include "muesli.h"

#define EPSILON 0.03
#define MAX_ITER 1000
#ifdef __HIPCC__
#define POW(a, b)      powf(a, b)
#define EXP(a)      exp(a)
#else
#define POW(a, b)      std::pow(a, b)
#define EXP(a)      std::exp(a)
#endif
int rows, cols;
int* input_image_int;
char* input_image_char;
bool ascii = false;

namespace msl {

    namespace jacobi {


        int readPGM(const std::string& filename, int& rows, int& cols, int& max_color)
        {
            std::ifstream ifs(filename, std::ios::binary);
            if (!ifs) {
                std::cout << "Error: Cannot open image file " << filename << "!" << std::endl;
                return 1;
            }

            // Read magic number.
            std::string magic;
            getline(ifs, magic);
            if (magic.compare("P5")) { // P5 is magic number for pgm binary format.
                if (magic.compare("P2")) { // P2 is magic number for pgm ascii format.
                    std::cout << "Error: Image not in PGM format!" << std::endl;
                    return 1;
                }
                ascii = true;
            }

            // Skip comments
            std::string inputLine;
            while (true) {
                getline(ifs, inputLine);
                if (inputLine[0] != '#') break;
            }

            // Read image size and max color.
            std::stringstream(inputLine) >> cols >> rows;
            getline(ifs, inputLine);
            std::stringstream(inputLine) >> max_color;

            // Read image.
            if (ascii) {
                input_image_int = new int[rows*cols];
                int i = 0;
                while (getline(ifs, inputLine)) {
                    std::stringstream(inputLine) >> input_image_int[i++];
                }
            } else {
                input_image_char = new char[rows*cols];
                ifs.read(input_image_char, rows*cols);
            }

            return 0;
        }

        int writePGM(const std::string& filename, DM<int>& out_image, int rows, int cols, int max_color)
        {
            std::ofstream ofs(filename, std::ios::binary);
            if (!ofs) {
                std::cout << "Error: Cannot open image file " << filename << "!" << std::endl;
                return 1;
            }

            // Gather full image
            int** img = new int*[rows];
            for (int i = 0; i < rows; i++)
                img[i] = new int[cols];

            // Write image header
            ofs << "P5\n" << cols << " " << rows << " " << std::endl << max_color << std::endl;

            // Write image
            for (int x = 0; x < rows; x++) {
                for (int y = 0; y < cols; y++) {
                    unsigned char intensity = static_cast<unsigned char> (img[x][y]);
                    ofs << intensity;
                }
            }

            if (ofs.fail()) {
                std::cout << "Cannot write file " << filename << "!" << std::endl;
                return 1;
            }

            return 0;
        }

        class GoLNeutralValueFunctor : public Functor2<int, int, int> {
        public:
            GoLNeutralValueFunctor(int default_neutral)
                    : default_neutral(default_neutral) {}

            MSL_USERFUNC
            int operator()(int x, int y) const {
                // All Border are not populated.
                return default_neutral;
            }

        private:
            int default_neutral = 0;
        };

/**
 * @brief Averages the top, bottom, left and right neighbours of a specific
 * element
 *
 */
        class Gaussian
                : public DMMapStencilFunctor<int, int, GoLNeutralValueFunctor> {
        public:
            Gaussian() : DMMapStencilFunctor(){}

            MSL_USERFUNC
            int operator() (int row, int col, PLMatrix<int> *input, int ncol, int nrow) const
            {
                int kw = 10;
                int offset = kw/2;
                float weight = 1.0f;
                float sigma = 1;
                float mean = (float)kw/2;

                // Convolution
                int sum = 0;
                for (int r = 0; r < kw; ++r) {
                    for (int c = 0; c < kw; ++c) {
                        sum += input->get(row+r-offset, col+c-offset) *
                               EXP(-0.5 * (POW((r-mean)/sigma, 2.0) + POW((c-mean)/sigma,2.0))) / (2 * M_PI * sigma * sigma);
                    }
                }

                return (int)sum/weight;
            }
        };


//         msl::jacobi::testGaussian(in_file, out_file, kw, output, tile_width, iterations, iterations_used);
        int testGaussian(std::string in_file, std::string out_file, int kw, bool output, int tile_width, int iterations, int iterations_used) {
            int max_color;
            double gauss_time = 0.0, t_upload = 0.0, t_padding =  0.0, t_kernel = 0.0;

            // Read image
            readPGM(in_file, rows, cols, max_color);
            msl::startTiming();
            for (int run = 0; run < Muesli::num_runs; ++run) {
                // Create distributed matrix to store the grey scale image.
                DM<int> gs_image(rows, cols, 1);
                DM<int> gs_image_result(rows, cols, 1);
                //writePGM("original.pgm", gs_image, rows, cols, max_color);

                double t = MPI_Wtime();
                // Gaussian blur
                //Gaussian g(kw);
                Gaussian g;

                g.setStencilSize(1);
                g.setTileWidth(tile_width);
                GoLNeutralValueFunctor dead_nvf(0);

                gs_image.mapStencilMM(gs_image_result, g, dead_nvf);
                //writePGM("afterGaussian.pgm", gs_image, rows, cols, max_color);

                // timing
                gauss_time += MPI_Wtime() - t;
                gauss_time = gauss_time;
            /*    t_upload += gs_image.getStencilTimes()[0];
                t_padding +=  gs_image.getStencilTimes()[1];
                t_kernel += gs_image.getStencilTimes()[2];*/

                if (output && msl::isRootProcess())
                    writePGM(out_file, gs_image, rows, cols, max_color);

                msl::splitTime(run);
            }
            msl::stopTiming();
            if (msl::isRootProcess()) {
                std::cout << "Gaussian time: " << gauss_time/Muesli::num_runs << std::endl
                          << "Upload time: " << t_upload/Muesli::num_runs << std::endl
                          << "Kernel time: " << t_kernel/Muesli::num_runs << std::endl
                          << "Padding time " << t_padding/Muesli::num_runs << std::endl;
            }
            return 0;
        }

    } // namespace jacobi
} // namespace msl

int init(int row, int col)
{
    if (ascii) return input_image_int[row*cols+col];
    else return input_image_char[row*cols+col];
}
int main(int argc, char **argv) {
    msl::initSkeletons(argc, argv);
    int nGPUs = 1;
    int nRuns = 1;
    int iterations = MAX_ITER;
    int tile_width = msl::DEFAULT_TILE_WIDTH;
    msl::Muesli::cpu_fraction = 0.0;
    //bool warmup = false;
    bool output = false;

    std::string in_file, out_file, file; //int kw = 10;
    file = "result.csv";
    if (argc >= 6) {
        nGPUs = atoi(argv[1]);
        nRuns = atoi(argv[2]);
        msl::Muesli::cpu_fraction = atof(argv[3]);
        if (msl::Muesli::cpu_fraction > 1) {
            msl::Muesli::cpu_fraction = 1;
        }
        tile_width = atoi(argv[4]);
        iterations = atoi(argv[5]);
    }
    if (argc == 7) {
        in_file = argv[8];
        size_t pos = in_file.find(".");
        out_file = in_file;
        out_file.insert(pos, "_gaussian");
    } else {
        in_file = "lena.pgm";
        out_file = "lena_gaussian.pgm";
        output = true;
        printf("I will take lena\n");
    }

    msl::setNumGpus(nGPUs);
    msl::setNumRuns(nRuns);

    int iterations_used=0;
    for (int r = 0; r < msl::Muesli::num_runs; ++r) {
        msl::jacobi::testGaussian(in_file, out_file, 10, output, tile_width, iterations, iterations_used);
    }

    if (output) {
/*        std::string id = "" + std::to_string(nGPUs) + ";" + std::to_string(tile_width) +";" + std::to_string(iterations) + ";" + std::to_string(iterations_used) +
                         ";" + std::to_string(msl::Muesli::cpu_fraction * 100) + ";\n";
        msl::printTimeToFile(id.c_str(), file);*/
        std::ofstream outputFile;
        outputFile.open(file, std::ios_base::app);
        outputFile << "" + std::to_string(nGPUs) + ";" + std::to_string(tile_width) +";" + std::to_string(iterations) + ";" + std::to_string(iterations_used) +
                      ";" + std::to_string(msl::Muesli::cpu_fraction * 100) + ";\n";
        outputFile.close();
    } else {
        msl::stopTiming();
    }
    msl::terminateSkeletons();
    return 0;
}
