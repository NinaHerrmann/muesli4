#include "hip/hip_runtime.h"

/*
 * map_kernels.cpp
 *
 *      Author: Steffen Ernsting <s.ernsting@uni-muenster.de>
 *
 * -------------------------------------------------------------------------------
 *
 * The MIT License
 *
 * Copyright 2014 Steffen Ernsting <s.ernsting@uni-muenster.de>,
 *                Herbert Kuchen <kuchen@uni-muenster.de.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

template <typename T, typename R, typename F>
__global__ void msl::detail::mapKernel(T *in, R *out, size_t size, F func) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < size) {
    out[x] = func(in[x]);
    //    printf("debug GPU: x: %i, in[x]: %i, out[x]: %i\n",x,in[x],out[x]);
  }
}

// new kernel for distributed matrices (DM)
template <typename T, typename R, typename F>
__global__ void msl::detail::mapIndexKernel(T *in, R *out, size_t size,
                                            size_t first, F func, int ncols) {
  size_t k = blockIdx.x * blockDim.x + threadIdx.x;
  int i = (k + first) / ncols;
  int j = (k + first) % ncols;
  if (k < size) {
    out[k] = func(i, j, in[k]);
  }
}

template <typename T, typename R, typename F>
__global__ void msl::detail::mapIndexKernel(T *in, R *out, size_t size,
                                            size_t first, F func,
                                            bool localIndices) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  size_t indexOffset = localIndices ? 0 : first;

  if (x < size) {
    out[x] = func(x + indexOffset, in[x]);
  }
}

template <typename T, typename R, typename F>
__global__ void msl::detail::mapIndexKernel(T *in, R *out,
                                            GPUExecutionPlan<T> plan, F func,
                                            bool localIndices) {
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  size_t rowOffset = localIndices ? 0 : plan.firstRow;
  size_t colOffset = localIndices ? 0 : plan.firstCol;

  if (y < plan.nLocal) {
    if (x < plan.mLocal) {
      out[y * plan.mLocal + x] =
          func(y + rowOffset, x + colOffset, in[y * plan.mLocal + x]);
    }
  }
}

template <typename T, typename R, typename F, typename NeutralValueFunctor>
__global__ void
msl::detail::mapStencilKernel(R *out, GPUExecutionPlan<T> plan,
                              PLMatrix<T> *input, F func,
                              int tile_width, int tile_height, NeutralValueFunctor nv) {

  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  input->readToSharedMem(y + plan.firstRow, x, tile_width, tile_height,
                         plan.gpuRows, plan.gpuCols);
  if (y < plan.gpuRows) {
    if (x < plan.gpuCols) {

      if (!((y == 0 && x < plan.firstCol) ||
            (y == (plan.gpuRows - 1) && x > plan.lastCol))) {
        out[y * plan.gpuCols + x - plan.firstCol] =
            func(y + plan.firstRow, x, *input);
      }
    }
  }
}
template <typename T, typename R, typename F, typename NeutralValueFunctor>
__global__ void
msl::detail::mapStencilMMKernel(R *out, GPUExecutionPlan<T> plan,
                                T *inputdm, T *inputpadding, F func,
                              int tile_width, int tile_height, NeutralValueFunctor nv) {

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ T inputsm[];


    // 512 threads per block --> assuming all in one line we need 512*3+2 numbers in shared memory = 1538 nicht so gut für die wiederverwendung
    // einen "Block" abarbeiten z.B. 16*16 = 256 Elemente 64 surrounding weil block 
    // approximately 16.000 floats per SM Palma
    // 0  1   2  3  4   5  6  7
    // 8  9  10 11 12 13 14 15
    // 16 17 18 19 20 21 22 23
    // 24 25 26 27 28 29 30 31
    // 32 33 34 35 36 37 38 39
    // 40 41 42 43 44 45 46 47
    // 48 49 50 51 52 53 54 55
    // 56 57 58 59 60 61 62 63
    // 0  1   2  3  4   5  6  7  8  9
    // 10 11 12 13 14  15 16 17 18 19
    // ...
    int localcol = (x - (blockIdx.x * tile_height));
    int localrow = y - (blockIdx.y * tile_width);
    int localindex = localrow * tile_width + localcol;
    int realdataoffset = tile_width + 3 + (localrow *  2);
    int secondrealdataoffset = y * plan.gpuCols + x - plan.firstCol;
    // Copy all the "real data"

    inputsm[localindex + realdataoffset] = __ldg(&inputdm[secondrealdataoffset]);

    // Copy borders
    int modulo = localindex % tile_width;

    int inputoffset = modulo + blockIdx.x * tile_width;
    if (blockIdx.y != 0) {
        // In case we are not the first block vertical copy top from previous gpu
        inputsm[modulo+1] =__ldg(&inputdm[((blockIdx.y) * (plan.gpuCols)) * tile_height - plan.gpuCols +
                inputoffset]);
    } else {
        // In case we are in the end blockIdx.y == 0 we need to copy the top from padding_stencil
        inputsm[modulo+1] = __ldg(&inputpadding[inputoffset]);
    }

    int bottomoffset = (tile_width + 2)*(tile_height +1)+1 + modulo;
    // Top done ... continue with bottom
    if (blockIdx.y == ((plan.gpuRows / tile_height)-1)){
        inputsm[bottomoffset] = __ldg(&inputpadding[inputoffset + plan.gpuCols]);
    } else {
        // In case it is not the last tile we need to copy bottom from top of other tile
        inputsm[bottomoffset] = __ldg(&inputdm[(blockIdx.y + 1) * (plan.gpuCols) * tile_height + inputoffset]);
    }

    if (localindex < tile_height) {
        int inputoffset = (blockIdx.y) * (plan.gpuCols) * tile_height + (plan.gpuCols * localindex);
        // Same for left and right, if blockidX.x is != 0 we need to copy from left
        // If blockidx is not the last block we need to copy from right
        int rightoffset = tile_width + 2 + localindex * (tile_width+2);
        if (blockIdx.x != 0) {
            inputsm[rightoffset] = __ldg(&inputdm[inputoffset +
                                         (blockIdx.x) * tile_width - 1]);
        } else {
            // In case we are in the ad blockIdx.y == 0 we need to copy from padding_stencil
            inputsm[rightoffset] = 100;
        }
        int leftoffset= tile_width+1 + (tile_width + 2) + localindex * (tile_width+2);
        // In case we are the last tile we need to copy from the stencil otherwise we copy from the other gpu
        if (blockIdx.x == ((plan.gpuCols / tile_width)-1)){
             inputsm[leftoffset] = 100;
        } else {
            // In case it is not the last tile we need to copy right hand side from other tile
            inputsm[leftoffset] =__ldg(&inputdm[inputoffset + (blockIdx.x +1) * tile_width]);
        }
    }
    __syncthreads();
    if (localrow < tile_height) {
        if (localcol < tile_width) {
            out[y * plan.gpuCols + x - plan.firstCol] = func(localrow, localcol, inputsm, tile_width, tile_height);
        }
    }

}
template <typename T> __global__ void msl::detail::printFromGPU(T *A, int size) {
  for (int i = 0; i < size; i++) {
      printf("[%.1f];", A[i]);
  }
}
