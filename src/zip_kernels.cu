#include "hip/hip_runtime.h"
/*
 * zip_kernels.cu
 *
 *      Authors: Steffen Ernsting <s.ernsting@uni-muenster.de>
 *               Herbert Kuchen <kuchen@uni-muenster.de.
 * 
 * -------------------------------------------------------------------------------
 *
 * The MIT License
 *
 * Copyright 2020 Steffen Ernsting <s.ernsting@uni-muenster.de>,
 *                Herbert Kuchen <kuchen@uni-muenster.de.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 *
 */

template <typename T1, typename T2, typename R, typename FCT2>
__global__ void msl::detail::zipKernel(T1* in1, T2* in2, R* out, size_t n, FCT2 func){
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n) {
    out[x] = func(in1[x], in2[x]);
  }
}

// new kernel for zip(InPlace)3, HK 19.11.2020
template <typename T1, typename T2, typename T3, typename R, typename FCT3>
__global__ void msl::detail::zipKernel(T1* in1, T2* in2, T3* in3, R* out, size_t n, FCT3 func){
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < n) {
    out[x] = func(in1[x], in2[x], in3[x]);
  }
}


// new kernel for DM, HK 06.11.2020 -- TODO better to start matrix of threads?
template <typename T1, typename T2, typename R, typename FCT3>
__global__ void msl::detail::zipIndexKernel(T1* in1,T2* in2,R* out,size_t n,int first,FCT3 func,int ncols){
  size_t k = blockIdx.x * blockDim.x + threadIdx.x;

  //if (k < n) {
  out[k] = func((k + first) / ncols,(k + first) % ncols, in1[k], in2[k]);
 // }
}

// new kernel for DM, NH 06.11.2020
template <typename T1, typename T2, typename R, typename FCT3>
__global__ void msl::detail::crossZipIndexKernel(T1* in1,T2* in2,R* out,size_t n,int first,FCT3 func,int ncols){
    size_t k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < n) {
        out[k] = func((k + first) / ncols,(k + first) % ncols, in1, in2);
    }
}
// new kernel for DM, NH 06.11.2020
template <typename T1, typename T2, typename FCT3>
__global__ void msl::detail::crossZipInPlaceIndexKernel(T1* in1,T2* in2,size_t n,int first,FCT3 func,int ncols){
    size_t k = blockIdx.x * blockDim.x + threadIdx.x;

    func((k + first) / ncols,(k + first) % ncols, in1, in2);

}
// new kernel for zipping a DM, two DAs and a DM, HK 20.11.2020
template <typename T1, typename T2, typename T3, typename T4, typename R, typename FCT3>
__global__ void msl::detail::zipKernelAAM(T1* in1, T2* in2, T3* in3, T4*in4,
                                          R* out, size_t n, int first, int first2, FCT3 func, int ncols){
  size_t k = blockIdx.x * blockDim.x + threadIdx.x;
  int i = ((k + first) / ncols) - first2;
  if (k < n) {
    out[k] = func(in1[k], in2[i], in3[i], in4[k]);
  }
}

template <typename T1, typename T2, typename R, typename FCT3>
__global__ void msl::detail::zipIndexKernel(T1* in1, T2* in2, R* out, size_t n,
                                            int first, FCT3 func, bool localIndices){
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = localIndices ? 0 : first;
  if (x < n) {
    out[x] = func(x + offset, in1[x], in2[x]);
  }
}

template <typename T1, typename T2, typename R, typename FCT4>
__global__ void msl::detail::zipIndexKernel(T1* in1, T2* in2, R* out, GPUExecutionPlan<T1> plan,
                                            FCT4 func,bool localIndices){
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  int rowOffset = localIndices ? 0 : plan.firstRow;
  int colOffset = localIndices ? 0 : plan.firstCol;

  if (y < plan.nLocal) {
    if (x < plan.mLocal) {
      out[y * plan.mLocal + x] = func(y + rowOffset,
    		     	 	 	 	 	  x + colOffset,
    		     	 	 	 	 	  in1[y * plan.mLocal + x],
    		     	 	 	 	 	  in2[y * plan.mLocal + x]);
    }
  }
}
template <typename T1, typename T2, typename R, typename FCT4>
__global__ void msl::detail::zipIndexKernel(T1* in1,T2* in2,R* out,GPUExecutionPlan<T1> plan,FCT4 func,
                                            bool localIndices, int nrow, int ncol, bool dim3){
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int rowOffset = localIndices ? 0 : plan.firstRow;
    int colOffset = localIndices ? 0 : plan.firstCol;
    int depthOffset = localIndices ? 0 : plan.firstDepth;

    //int overall = ((z+depthOffset) * (nrow*ncol)) + (y * ncol) + x;
    int localoverall = (z * (nrow*ncol)) + (y * ncol) + x;
    if (z < plan.gpuDepth) {
        if (y < plan.gpuRows) {
            if (x < plan.gpuCols) {
                out[localoverall] = func(y + rowOffset,
                                         x + colOffset,
                                         z + depthOffset,
                                                in1[localoverall],
                                                in2[localoverall]);
            }
    }
  }
}






